#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

__global__
void saxpy(int n, float a, float *x, float *y){
          // n length, a scalar, x and y array pointers

  // for (int IDX = blockDim.x * blockIdx.x + threadIdx.x; 
  //     IDX<n; 
  //     IDX += blockDim.x * gridDim.x ){
  //
  //   y[IDX] = a*x[IDX] + y[IDX];
  // }


  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<n) y[idx] = a*x[idx]+y[idx];
}

// main
int main(void){


  // Parameters
  int N     =  1<<20;
  float a   = 2.0f;
  int bsize = 256;
  int gsize = (N+bsize-1)/bsize;


  // declare array pointers in host and device
  float *x, *y, *dev_x, *dev_y;

  // host allocate
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  // allocate in device // in unified memory, we would use hipMallocManaged()
  // at addres of dev_x, size
  hipMalloc(&dev_x, N*sizeof(float));
  hipMalloc(&dev_y, N*sizeof(float));

  // populate host arrays
  for (int idx=0; idx<N; idx++){
    x[idx] = 1.0f;
    y[idx] = 2.0f;    
  }

  // copy to device memory
  // https://developer.download.nvidia.com/compute/DevZone/docs/html/C/doc/html/group__CUDART__MEMORY_g48efa06b81cc031b2aa6fdc2e9930741.html
  // destination, source, byteCount, typeTransfer
  hipError_t err = hipMemcpy(dev_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  if (err!=hipSuccess){
    std::cerr<<"cudaMemError: "<<hipGetErrorString(err)<<std::endl;
    return -1;
  }
  err = hipMemcpy(dev_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess){
    std::cerr<<"cudaMemError: "<< hipGetErrorString(err)<< std::endl;
    return -1;
  }


  // we have run error checks after each device-memory assignation


  // now run the SAXPY kernel
  // num elem, alpha scalar, device array 1, device array 2
  saxpy<<<gsize,bsize>>>(N,a, dev_x, dev_y);
  err = hipGetLastError();
  if (err != hipSuccess){
    std::cerr<<"Kernel launch err: "<<hipGetErrorString(err)<<std::endl;
    return -1;
  }


  // copy dev memory back to host
  err = hipMemcpy(y, dev_y, N*sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess){
    std::cerr<<"Mem erro dev2host: "<<hipGetErrorString(err)<<std::endl;
    return -1;
  }

  // numerical error checks
  float maxerr = 0.0f;
  float accumerr = 0.0f;
  for (int idx = 0; idx<N; idx++){
    accumerr = accumerr +fabs(y[idx]-4.0f);
    maxerr = max(maxerr , abs(y[idx]-4.0f));
  }
  printf("N: %d; gsize: %d; bsize: %d\n", N, gsize, bsize);
  printf("Max err: %f ; Accum error: %f\n", maxerr, accumerr);



  hipFree(dev_x);
  hipFree(dev_y);
  free(x);
  free(y);


}
