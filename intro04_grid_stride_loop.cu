#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>


__global__

void saxpy(int n, float a, float * x, float* y){
    //int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx<n){
    //    y[idx] = a*x[idx] + y[idx];
    //}

    // which can be expressed better:

    for (int idx = blockDim.x * blockIdx.x + threadIdx.x ;
            idx < n;
            idx += blockDim.x * gridDim.x) // increase by stride
    {
        y[idx] = a * x[idx] + y[idx];
    }
}



int main(void){

    int SMs; // stream multiprocessors
    //char hipDeviceAttributeMultiprocessorCount;
    //hipDeviceGetAttribute(&SMs,0,0);
    int devId;
    hipDeviceGetAttribute(&SMs, hipDeviceAttributeMaxGridDimX, &devId);



    int N = 10000;
    float a=3.0, val1=1.0, val2=2.0;

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // populate

    for (int jdx=0; jdx<N; jdx++){
        x[jdx] = val1;
        y[jdx] = val2;
    }

    // operate saxpy
    saxpy<<<4096,256>>>(N, a, x, y);
    // error check
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess){
        std::cerr << "Cuda synch err: "<< hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipGetLastError();
    if (err!=hipSuccess){
        std::cerr << "Cuda kernel launch error: "<<hipGetErrorString(err)<< std::endl;
        return -1;
    }

    // print out values
    //for (int idx=0; idx<N; idx++){
    //    printf("%1.2f ",y[idx]);
    //}

    float maxerr = 0.0f;
    float total = a*val1 + val2;
    for (int idx=0; idx<N;idx++)
    {
        maxerr = fmax(maxerr, fabs(y[idx]-total));
    }
    std::cout << "Max err: "<< maxerr<<std::endl;

    return EXIT_SUCCESS;
}

